#include "hip/hip_runtime.h"
#include "reduction.cuh"

// naive reduction kernel
// This kernel is barely optimized and serves as a baseline
__global__ void reductionNaive(int *output, const int *input,
                               const int length) {
    int i = blockIdx.x * NAIVE_BLOCK_DIM;
    int partialSum = 0;
    
    for (int j = 0; j < NAIVE_BLOCK_DIM; j++) {
        if (i + j < length) {
            partialSum += input[i + j];
        }
    }
    atomicAdd(output, partialSum);
}

#define LIMITED_KERNEL_LOOP(x, n, step) \
    for (x = (threadIdx.x + blockIdx.x * blockDim.x) * step; x < n; x += step * (gridDim.x * blockDim.x))

__device__ int warpSumReduce(int val) {
    unsigned mask = 0xFFFFFFFF; 
    for (int offset = 16; offset > 0; offset >>= 1) {
        val += __shfl_down_sync(mask, val, offset);
    }
    return val;
}

// Your optimized kernel implementations go here
__global__ void reductionOptimized(int *output, const int *input, const int length) {
    __shared__ int smem[OPTIM_BLOCK_SIZE];
    int4 partialSum = {0, 0, 0, 0};
    int x;
    LIMITED_KERNEL_LOOP(x, length - OPTIM_HANDLE_NUM, OPTIM_HANDLE_NUM) {
        partialSum.x += input[x];
        partialSum.y += input[x + 1];
        partialSum.z += input[x + 2];
        partialSum.w += input[x + 3];
    }
    int sum = partialSum.x + partialSum.y + partialSum.z + partialSum.w;
    for(; x < length; x++) sum += input[x];
    sum = warpSumReduce(sum);
    if (threadIdx.x % WARP_SIZE == 0) smem[threadIdx.x / WARP_SIZE] = sum;
    __syncthreads();
    if (threadIdx.x < WARP_SIZE) {
        sum = threadIdx.x < OPTIM_BLOCK_SIZE / WARP_SIZE ? smem[threadIdx.x] : 0;
        sum = warpSumReduce(sum);
        if (threadIdx.x == 0) atomicAdd(output, sum);
    }
}